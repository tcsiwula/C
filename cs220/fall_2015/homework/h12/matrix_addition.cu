#include "hip/hip_runtime.h"
/* File:     matrix_addition.cu
 * Purpose:  Implement vector addition on a gpu using cuda
 *
 * Compile:  nvcc [-g] [-G] -arch=sm_21 -o vec_add vec_add.cu
 * Run:      ./vec_add <n>
 *              n is the vector length
 *
 * Input:    None
 * Output:   Result of vector addition.  If all goes well it should
 *           be a vector consisting of n copies of n+1.
 *
 * Notes:
 * 1.  CUDA is installed on all of the machines in HR 530, HR 235, and
 *     and LS G12
 * 2.  If you get something like "nvcc: command not found" when you try
 *     to compile your program.  Type the following command
 *
 *           $ export PATH=/usr/local/cuda/bin:$PATH
 *
 *     (As usual the "$" is the shell prompt:  just type the rest
 *     of the line.)
 */
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

/* Kernel for vector addition */
__global__ void Mat_add(float A[], float B[], float C[], int m, int n) {
   /* blockDim.x = threads_per_block                            */
   /* First block gets first threads_per_block components.      */
   /* Second block gets next threads_per_block components, etc. */
   int my_ij = blockDim.x * blockIdx.x + threadIdx.x;

   /* The test shouldn't be necessary */
   if (blockIdx.x < m && threadIdx.x < n)
      C[my_ij] = A[my_ij] + B[my_ij];
}

/* Host code */
int main(int argc, char* argv[]) {
   int n, i, m;
   float *h_x, *h_y, *h_z;
   float *d_x, *d_y, *d_z;
   int threads_per_block;
   int block_count;
   size_t size;

   /* Get number of components in vector */
   if (argc != 2) {
      fprintf(stderr, "usage: %s <vector order>\n", argv[0]);
      exit(0);
   }
   m = strtol(argv[1], NULL, 10);
   n = strtol(argv[1], NULL, 10);
   printf("m = %d, n = %d\n", m, n);
   size = m*n*sizeof(float);

   /* Allocate input vectors in host memory */
   h_x = (float*) malloc(size);
   h_y = (float*) malloc(size);
   h_z = (float*) malloc(size);

    printf("Enter the matrices A and B\n");
    Read_matrix(h_x, m, n);
    Read_matrix(h_y, m, n);

     Print_matrix("A =", h_x, m, n);
     Print_matrix("B =", h_y, m, n);

   /* Allocate vectors in device memory */
   hipMalloc(&d_x, size);
   hipMalloc(&d_y, size);
   hipMalloc(&d_z, size);

   /* Copy vectors from host memory to device memory */
   hipMemcpy(d_x, h_x, size, hipMemcpyHostToDevice);
   hipMemcpy(d_y, h_y, size, hipMemcpyHostToDevice);


   /* Invoke kernel using block_count blocks, each of which  */
   /* contains threads_per_block threads                     */
   Mat_add<<<i, n>>>(d_A, d_B, d_C, m, n);

   /* Wait for the kernel to complete */
   hipDeviceSynchronize();

   /* Copy result from device memory to host memory */
   /* h_z contains the result in host memory        */
   hipMemcpy(h_z, d_z, size, hipMemcpyDeviceToHost);


   Print_matrix("The sum is: ", h_z, m, n);

   /* Free device memory */
   hipFree(d_x);
   hipFree(d_y);
   hipFree(d_z);

   /* Free host memory */
   free(h_x);
   free(h_y);
   free(h_z);

   return 0;
}  /* main */

void Read_matrix(float A[], int m, int n) {
   int i, j;

   for (i = 0; i < m; i++)
      for (j = 0; j < n; j++)
         scanf("%f", &A[i*n+j]);
}

void Print_matrix(char title[], float A[], int m, int n) {
   int i, j;

   printf("%s\n", title);
   for (i = 0; i < m; i++) {
      for (j = 0; j < n; j++)
         printf("%.1f ", A[i*n+j]);
      printf("\n");
   }
}
