#include "hip/hip_runtime.h"
/* File:     integral.cu
    Author:  Tim Siwula
 * Purpose:  Implement dot product on a gpu using cuda.  This version
 *           uses a binary tree reduction in which we attempt to reduce
 *           thread divergence.  It also uses shared memory to store
 *           intermediate results.  Assumes both threads_per_block and
 *           blocks_per_grid are powers of 2.
 *
 * Compile:  nvcc -arch=sm_21 -o integral integral.cu
 * Run:      ./integral <n> <a> <b> <blocks> <threads_per_block>
 *              n is the vector length
 *              i.e: ./integral 16 5 1 4 4
 * Input:    None
 * Output:   Result of dot product of a collection of random floats
 *
 */
#include <stdio.h>
#include <stdlib.h>
#include "timer.h"

#define MAX_BLOCK_SZ 512

__device__ float Dev_f(float x) {
         return x*x + 1;
      }

/*-------------------------------------------------------------------
 * Function:    Dev_dot  (kernel)
 * Purpose:     Implement a dot product of floating point vectors
 *              using atomic operations for the global sum
 * In args:     x, y, n
 * Out arg:     z
 *
 */
__global__ void Trap(float a[], float b[], float total[], int n) {
   /* Use tmp to store products of vector components in each block */
   /* Can't use variable dimension here                            */
   __shared__ float block_sum[MAX_BLOCK_SZ];
   int t = blockDim.x * blockIdx.x + threadIdx.x;
   int loc_t = threadIdx.x;
   float h = (b[t]-a[t])/n;
   int left = a[t] + (t * h);
   float right = left + h;
   float tmp_left = Dev_f(left);
   float tmp_right = Dev_f(right);
   float tmp_area = h * (tmp_left + tmp_right);
   float area = .5 * tmp_area;

   if (t < n) block_sum[loc_t] = area;
   __syncthreads();


   /* This uses a tree structure to do the addtions */
   for (int stride = blockDim.x/2; stride >  0; stride /= 2) {
      if (loc_t < stride)
         block_sum[loc_t] += block_sum[loc_t + stride];
      __syncthreads();
   }

   /* Store the result from this cache block in z[blockIdx.x] */
   if (threadIdx.x == 0) {
      total[blockIdx.x] = block_sum[0];
   }
}  /* Dev_dot */

/*-------------------------------------------------------------------
 * Function:  Dot_wrapper
 * Purpose:   CPU wrapper function for GPU dot product
 * Note:      Assumes x_d, y_d have already been
 *            allocated and initialized on device.  Also
 *            assumes z_d has been allocated.
 */
float Dot_wrapper(float a_d[], float b_d[], float total_d[],
      int n, int blocks, int threads) {
   int i;
   float integral = 0.0;
   float total_h[blocks];

   /* Invoke kernel */
   Trap<<<blocks, threads>>>(a_d, b_d, total_d, n);
   hipDeviceSynchronize();

   hipMemcpy(total_h, total_d, blocks*sizeof(float), hipMemcpyDeviceToHost);

   for (i = 0; i < blocks; i++)
      integral += total_h[i];
   return integral;
}  /* Dot_wrapper */


/*-------------------------------------------------------------------
 * Host code
 */
void Get_args(int argc, char* argv[], int* n_p, int* a_p, int* b_p, int* threads_per_block_p,
      int* blocks_per_grid_p);
void Setup(int n, int a, int b, int blocks, float** x_h_p, float** y_h_p, float** x_d_p,
      float** y_d_p, float** z_d_p);
float Serial_dot(float x[], float y[], int n);
void Free_mem(float* x_h, float* y_h, float* x_d, float* y_d,
      float* z_d);
float Dot_wrapper(float x_d[], float y_d[], float z_d[],
      int n, int blocks, int threads);

/*-------------------------------------------------------------------
 * main
 */
int main(int argc, char* argv[]) {
   int n, a, b, threads_per_block, blocks_per_grid;
   float *a_h, *b_h, integral = 0;
   float *a_d, *b_d, *total_d;
   double start, finish;  /* Only used on host */

   Get_args(argc, argv, &n, &a, &b, &threads_per_block, &blocks_per_grid);
   Setup(n, a, b, blocks_per_grid, &a_h, &b_h, &a_d, &b_d, &total_d);

   GET_TIME(start);
   integral = Dot_wrapper(a_d, b_d, total_d, n, blocks_per_grid,
         threads_per_block);
   GET_TIME(finish);

   printf("The integral as computed by cuda is: %e\n", integral);
   printf("Elapsed time for cuda = %e seconds\n", finish-start);


   Free_mem(a_h, b_h, total_d, b_d, total_d);

   return 0;
}  /* main */


/*-------------------------------------------------------------------
 * Function:  Get_args
 * Purpose:   Get and check command line args.  If there's an error
 *            quit.
 */
void Get_args(int argc, char* argv[], int* n_p, int* a_p, int* b_p, int* threads_per_block_p,
      int* blocks_per_grid_p) {

   if (argc != 6) {
      fprintf(stderr, "usage: %s <n> <a> <b> <blocks> <threads>\n",
            argv[0]);
      exit(0);
   }
   *n_p = strtol(argv[1], NULL, 10);
   *a_p = strtol(argv[2], NULL, 10);
   *b_p = strtol(argv[3], NULL, 10);
   *blocks_per_grid_p = strtol(argv[4], NULL, 10);
   *threads_per_block_p = strtol(argv[5], NULL, 10);
}  /* Get_args */

/*-------------------------------------------------------------------
 * Function:  Setup
 * Purpose:   Allocate and initialize host and device memory
 */
void Setup(int n, int a, int b, int blocks, float** a_h_p, float** b_h_p, float** a_d_p,
      float** b_d_p, float** total_d_p) {
   int i;
   size_t size = n*sizeof(float);

   /* Allocate input vectors in host memory */
   *a_h_p = (float*) malloc(size);
   *b_h_p = (float*) malloc(size);

   /* Initialize input vectors */
   srandom(1);
   for (i = 0; i < n; i++) {
      (*a_h_p)[i] = random()/((double) RAND_MAX);
      (*b_h_p)[i] = random()/((double) RAND_MAX);
   }

   /* Allocate vectors in device memory */
   hipMalloc(a_d_p, size);
   hipMalloc(a_d_p, size);
   hipMalloc(total_d_p, blocks*sizeof(float));

   /* Copy vectors from host memory to device memory */
   hipMemcpy(*a_d_p, *a_h_p, size, hipMemcpyHostToDevice);
   hipMemcpy(*b_d_p, *b_h_p, size, hipMemcpyHostToDevice);
}  /* Setup */

/*-------------------------------------------------------------------
 * Function:  Serial_dot
 * Purpose:   Compute a dot product on the cpu
 */
float Serial_dot(float x[], float y[], int n) {
   int i;
   float dot = 0;

   for (i = 0; i < n; i++)
      dot += x[i]*y[i];

   return dot;
}  /* Serial_dot */

/*-------------------------------------------------------------------
 * Function:  Free_mem
 * Purpose:   Free host and device memory
 */
void Free_mem(float* x_h, float* y_h, float* x_d, float* y_d,
      float* z_d) {

   /* Free device memory */
   hipFree(x_d);
   hipFree(y_d);
   hipFree(z_d);

   /* Free host memory */
   free(x_h);
   free(y_h);

}  /* Free_mem */
